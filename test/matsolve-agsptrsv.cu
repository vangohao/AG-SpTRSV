#include "hip/hip_runtime.h"
// ref:
// https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSPARSE/spsv_csr/spsv_csr_example.c

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include <fstream>
#include <iostream>
#include <string>

#include "unisolver/ArrayUtils.hpp"
#include "unisolver/JsonUtils.hpp"

#include "AG-SpTRSV.h"

#include "utils.h"

#include "YYSpTRSV.h"

#include "spts_syncfree_cuda.h"

using namespace uni;

#define VALUE_TYPE double
#define VALUE_SIZE 8

#define REPEAT_TIME 11
#define WARM_UP 1

#define ag_duration(a, b) \
    (1.0 * (b.tv_usec - a.tv_usec + (b.tv_sec - a.tv_sec) * 1.0e6))

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess) {                                   \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            while (1);                                                 \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS) {                           \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            while (1);                                                     \
        }                                                                  \
    }

using cusp_int = int;
#define my_CUSPARSE_INDEX HIPSPARSE_INDEX_32I

#define MAX_DOF_TEST 8

struct benchmark_record {
    double total_time = 0;
    long flops = 0;
    long bytes = 0;
    long count = 0;
};

benchmark_record benchmark_record_map_lower[MAX_DOF_TEST];

void RunBenchmarkLowerWithCusparse(Json json, int Dof, int stencil_type,
                                   int stencil_width) {
    constexpr int Dim = 3;

    std::string dof_str = std::to_string(Dof);
    cusp_int M = json[dof_str]["M"].get<cusp_int>();
    cusp_int N = json[dof_str]["N"].get<cusp_int>();
    cusp_int P = json[dof_str]["P"].get<cusp_int>();

    std::vector<std::array<cusp_int, Dim>> stencil_points;
    if (stencil_type == 0) {
        for (int d = Dim - 1; d >= 0; d--) {
            for (int j = stencil_width; j > 0; j--) {
                std::array<cusp_int, Dim> pt = {0, 0, 0};
                pt[d] = -j;
                stencil_points.push_back(pt);
            }
        }
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, 0});
    } else if (stencil_type == 1) {
        NestedLoop(
            constant_array<cusp_int, Dim>(-stencil_width),
            constant_array<cusp_int, Dim>(2 * stencil_width + 1), [&](auto pt) {
                cusp_int cnt = CartToFlat(
                    pt + stencil_width,
                    constant_array<cusp_int, Dim>(2 * stencil_width + 1));
                if (cnt < (myPow(2 * stencil_width + 1, Dim) / 2)) {
                    stencil_points.push_back(pt);
                }
            });
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, 0});
    } else {
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, -1});
        stencil_points.push_back(std::array<cusp_int, Dim>{1, 0, -1});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 1, -1});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, -1, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{1, -1, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{-1, 0, 0});
        stencil_points.push_back(std::array<cusp_int, Dim>{0, 0, 0});
    }

    // Host problem definition
    cusp_int A_num_rows = M * N * P * Dof;
    cusp_int A_nnz = 0;
    std::vector<cusp_int> hA_csrOffsets;
    std::vector<cusp_int> hA_columns;
    std::vector<double> hA_values;
    std::vector<double> hX;
    std::vector<double> hY;
    std::vector<double> hY_result;
    // 注意这里求解的是A* Y = X, 所以这里的Y是输出, X是输入

    // set A & hX
    NestedLoop(
        std::array<cusp_int, Dim>{}, std::array<cusp_int, Dim>{M, N, P},
        [&](auto loc) {
            for (int d = 0; d < Dof; d++) {
                hA_csrOffsets.push_back(A_nnz);
                cusp_int cnt = 0;
                for (auto pt : stencil_points) {
                    if (in_range(loc + pt, std::array<cusp_int, Dim>{},
                                 std::array<cusp_int, Dim>{M, N, P} - 1)) {
                        for (int k = 0; k < Dof; k++) {
                            if (pt != std::array<cusp_int, Dim>{0, 0, 0} ||
                                k == d) {
                                hA_columns.push_back(
                                    CartToFlat(
                                        loc + pt,
                                        std::array<cusp_int, Dim>{M, N, P}) *
                                        Dof +
                                    k);
                                hA_values.push_back(1.);
                                A_nnz++;
                                cnt++;
                            }
                        }
                    }
                }
                hX.push_back(cnt);
            }
        });
    hA_csrOffsets.push_back(A_nnz);

    std::cout << "A_nnz = " << A_nnz << "\n";

    // set hY
    hY.resize(A_num_rows);
    hY_result.resize(A_num_rows);
    for (cusp_int i = 0; i < A_num_rows; i++) hY_result[i] = 1.0;

    //--------------------------------------------------------------------------

    /* !!!!!! start computing SpTRSV !!!!!!!! */

    struct timeval tv_begin, tv_end;

    gettimeofday(&tv_begin, NULL);

    PREPROCESSING_STRATEGY ps = ROW_BLOCK;
    SCHEDULE_STRATEGY strategy = SIMPLE;
    int rb = 1;

    // int graph_reorder = 0;

    ptr_handler handler;

    // if (graph_reorder) {
    //     printf("Begin reordering\n");

    //     handler =
    //         SpTRSV_preprocessing(A_num_rows, A_nnz, hA_csrOffsets.data(),
    //         hA_columns.data(), ROW_BLOCK, 1);

    //     graph_reorder_with_level(handler);

    //     int permutation[A_num_rows];

    //     matrix_reorder(handler, permutation, hA_csrOffsets.data(),
    //     hA_columns.data(), hA_values.data());

    //     graph_finalize(handler);
    // }

    int flag;
    float sptrsv_time = 0;

    flag = 1;

    // matrix examples
    // atmosmodd
    // anaparas paras = anaparas(1024, 4, ROW_BLOCK, 1, LEVEL_WISE, ONE_LEVEL,
    // SIMPLE, RG_SIMPLE);

    // Wiki-Talk
    // anaparas paras = anaparas(64, 1, ROW_BLOCK, 1, LEVEL_WISE, ONE_LEVEL,
    // WORKLOAD_BALANCE, RG_SIMPLE);

    // cant
    // anaparas paras = anaparas(64, 2, ROW_BLOCK, 1, LEVEL_WISE, ONE_LEVEL,
    // SIMPLE2, RG_SIMPLE);

    // delaunay_n23
    // anaparas paras = anaparas(1024, 4, ROW_BLOCK_AVG, 2, LEVEL_WISE,
    // THRESH_LEVEL, SIMPLE, RG_SIMPLE);

    // delaunay_n13
    // anaparas paras = anaparas(64, 4, ROW_BLOCK_AVG, 1, LEVEL_WISE,
    // THRESH_LEVEL, WORKLOAD_BALANCE, RG_SIMPLE);

    // webbase-1M
    // anaparas paras = anaparas(1024, 1, ROW_BLOCK_THRESH, 32, LEVEL_WISE,
    // ONE_LEVEL, SIMPLE, RG_SIMPLE);

    // my 384**3 d3n4 stencilstar with diag
    // tbs 1024 sws 4 ps 0 8 0 ss 1 0 1
    // 前面都按照顺序，最后三个的顺序是反的
    // anaparas paras = anaparas(1024, 4, ROW_BLOCK, 8, LEVEL_WISE, ONE_LEVEL,
    //                           SIMPLE, RG_BALANCE);

    // my 512**3 d3n4 stencilstar with diag
    // tbs 1024 sws 4 ps 0 16 0 ss 0 0 1
    anaparas paras = anaparas(1024, 4, ROW_BLOCK, 16, LEVEL_WISE, ONE_LEVEL,
                              SIMPLE, RG_SIMPLE);

    // my 320**3 d3n14 stencilbox with diag
    // tbs 1024 sws 4 ps 0 4 0 ss 0 1 1
    // anaparas paras = anaparas(1024, 4, ROW_BLOCK, 4, LEVEL_WISE, ONE_LEVEL,
    // SIMPLE2, RG_SIMPLE);

    // my 384**3 d3n7 stencilstar width = 2 with diag
    // tbs 1024 sws 1 ps 1 16 0 ss 0 0 1
    // anaparas paras = anaparas(1024, 1, ROW_BLOCK_THRESH, 16, LEVEL_WISE,
    // ONE_LEVEL, SIMPLE, RG_SIMPLE);

    // my 384**3 stencilstar width = 1 fill in level =1 with diag
    // tbs 1024 sws 4 ps 0 8 0 ss 0 0 1
    // anaparas paras = anaparas(1024, 4, ROW_BLOCK, 8, LEVEL_WISE, ONE_LEVEL,
    // SIMPLE, RG_SIMPLE);

    show_paras(paras);

    ptr_anainfo ana = new anainfo(A_num_rows);
    SpTRSV_preprocessing_new(A_num_rows, A_nnz, hA_csrOffsets.data(),
                             hA_columns.data(), ana, paras);

    gettimeofday(&tv_end, NULL);

    printf("Preprocessing time: %.2f us\n", ag_duration(tv_begin, tv_end));

    // copy matrix and vector from CPU to GPU memory
    int *csrRowPtr_d, *csrColIdx_d;
    VALUE_TYPE *csrValue_d, *b_d, *x_d;
    hipMalloc(&csrRowPtr_d, sizeof(int) * (A_num_rows + 1));
    hipMemcpy(csrRowPtr_d, hA_csrOffsets.data(),
               sizeof(int) * (A_num_rows + 1), hipMemcpyHostToDevice);
    hipMalloc(&csrColIdx_d, sizeof(int) * A_nnz);
    hipMemcpy(csrColIdx_d, hA_columns.data(), sizeof(int) * A_nnz,
               hipMemcpyHostToDevice);
    hipMalloc(&csrValue_d, sizeof(VALUE_TYPE) * A_nnz);
    hipMemcpy(csrValue_d, hA_values.data(), sizeof(VALUE_TYPE) * A_nnz,
               hipMemcpyHostToDevice);
    hipMalloc(&b_d, sizeof(VALUE_TYPE) * A_num_rows);
    hipMemcpy(b_d, hX.data(), sizeof(VALUE_TYPE) * A_num_rows,
               hipMemcpyHostToDevice);
    hipMalloc(&x_d, sizeof(VALUE_TYPE) * A_num_rows);
    hipMemset(x_d, 0, sizeof(VALUE_TYPE) * A_num_rows);

    for (int i = 0; i < REPEAT_TIME; i++) {
        hipMemset(ana->get_value, 0, sizeof(int) * A_num_rows);
        hipMemset(x_d, 0, sizeof(VALUE_TYPE) * A_num_rows);

        hipDeviceSynchronize();

        gettimeofday(&tv_begin, NULL);

        SpTRSV_executor_variant(ana, paras, csrRowPtr_d, csrColIdx_d,
                                csrValue_d, b_d, x_d);
        hipDeviceSynchronize();

        gettimeofday(&tv_end, NULL);

        if (i >= WARM_UP) sptrsv_time += ag_duration(tv_begin, tv_end);
    }

    sptrsv_time *= 1e-6;

    hipMemcpy(hY.data(), x_d, sizeof(VALUE_TYPE) * A_num_rows,
               hipMemcpyDeviceToHost);

    //--------------------------------------------------------------------------

    long readBytes = (sizeof(cusp_int) + sizeof(double)) * A_nnz +
                     sizeof(cusp_int) * A_num_rows +
                     sizeof(double) * A_num_rows;
    long writeBytes = sizeof(double) * A_num_rows;

    benchmark_record_map_lower[Dof - 1] = {
        sptrsv_time, 2L * A_nnz * (REPEAT_TIME - WARM_UP),
        (readBytes + writeBytes) * (REPEAT_TIME - WARM_UP),
        (REPEAT_TIME - WARM_UP)};
    std::cout << "agsptrsv (10 runs) LowerTime(ms): " << sptrsv_time
              << ", Gflops: "
              << (2L * A_nnz * (REPEAT_TIME - WARM_UP) / sptrsv_time) * 1e-9
              << ", Bandwidth="
              << ((readBytes + writeBytes) * (REPEAT_TIME - WARM_UP) /
                  sptrsv_time) *
                     1e-9
              << std::endl;

    //--------------------------------------------------------------------------
    // device result check

    int correct = 1;
    for (cusp_int i = 0; i < A_num_rows; i++) {
        if (hY[i] !=
            hY_result[i]) {  // direct doubleing point comparison is not
            correct = 0;     // reliable
            // break;
            std::cout << "i = " << i << ", hY[i] = " << hY[i]
                      << ", hY_result[i] = " << hY_result[i] << std::endl;
        }
    }
    if (correct)
        printf("agsptrsv test PASSED\n");
    else
        printf("agsptrsv test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    // log::FunctionEnd(0, 0, 0);
    hipFree(csrRowPtr_d);
    hipFree(csrColIdx_d);
    hipFree(csrValue_d);
    hipFree(x_d);
    hipFree(b_d);
}

int main(int argc, char **argv) {
    Json json = LoadJsonFromFile("matsolve-csrgen.json");
    std::string problems[] = {"stencilstar", "stencilbox", "stencilstarfill1"};
    bool if_output = json["output"];
    for (int i = 0; i < 3; i++) {
        for (int stencil_width_0 = 0; stencil_width_0 < (i == 0 ? 2 : 1);
             stencil_width_0++) {
            int stencil_width = stencil_width_0 + 1;
            std::string problem = problems[i];

            std::ofstream of;
            if (if_output) {
                of.open(std::string{"results/matsolve-agsptrsv-"} + problem +
                        "-stencilwidth" + std::to_string(stencil_width) +
                        ".out");
            } else {
                of.open("/dev/null");
            }
            for (int dof = 0; dof < MAX_DOF_TEST; dof++) {
                of << problem << ", width=" << stencil_width
                   << ", dof=" << dof + 1 << std::endl;
                RunBenchmarkLowerWithCusparse(
                    json[problem + std::to_string(stencil_width)], dof + 1, i,
                    stencil_width);
                of << "Lower:";
                double total_time = benchmark_record_map_lower[dof].total_time;
                double total_flops_time =
                    static_cast<double>(benchmark_record_map_lower[dof].flops) /
                    total_time;
                double total_bytes_time =
                    static_cast<double>(benchmark_record_map_lower[dof].bytes) /
                    total_time;

                of << dof + 1 << "," << total_time << ","
                   << total_flops_time * 1e-9 << "," << total_bytes_time * 1e-9
                   << std::endl;
            }
            of.close();
        }
    }
    return 0;
}
