#include "hip/hip_runtime.h"
#include "AG-SpTRSV.h"
#include "utils.h"
#include "YYSpTRSV.h"
#include "spts_syncfree_cuda.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <unistd.h>

#define VALUE_TYPE double
#define VALUE_SIZE 8

#define ERROR_THRESH 1e-4

// #define REPEAT_TIME 100
// #define WARM_UP 10

#define REPEAT_TIME 10
#define WARM_UP 1

#define CU_TEST false

#define duration(a, b) (1.0 * (b.tv_usec - a.tv_usec + (b.tv_sec - a.tv_sec) * 1.0e6))

int error_detect(VALUE_TYPE *x, VALUE_TYPE *x_base, int m)
{
    VALUE_TYPE max_error = 0;
    int maxi = -1;
    for (int i = 0; i < m; i++)
    {
        if (fabs(x[i] - x_base[i]) > max_error)
        {
            max_error = fabs(x[i] - x_base[i]);
            maxi = i;
        }
    }
    return maxi;
}

int main(int argc, char* argv[])
{
    // hipSetDevice(1);

    struct timeval tv_begin, tv_end;

    int ch;

    int input_flag = 0, graph_flag = 0, outcsv_flag = 0;
    char *input_name, *graph_name, *outcsv_name;

    while ((ch = getopt(argc, argv, "g:o:i:")) != -1)
    {
        switch (ch)
        {
            case 'g':
                graph_flag = 1;
                graph_name = optarg;
                break;
            
            case 'o':
                outcsv_flag = 1;
                outcsv_name = optarg;
                break;

            case 'i':
                input_flag = 1;
                input_name = optarg;
                break;
        }
    }

    if (input_flag == 0)
    {
        printf("[Usage]: ./main_batch -i {input_filename}\n");
        exit(1);
    }

    // Original matrix A;
    int m;
    // int n;
    // int nnzA;
    // int *csrRowPtrA;
    // int *csrColIdxA;
    // VALUE_TYPE *csrValA;

    //read_mtx(argv[1], &m, &n, &nnzA, &csrRowPtrA, &csrColIdxA, &csrValA);

    // Triangular matrix L;
    int nnzL;
    int *csrRowPtrL;
    int *csrColIdxL;
    VALUE_TYPE *csrValL;

    printf("Start Read matrix!\n");

    read_tri<VALUE_TYPE>(input_name, &m, &nnzL, &csrRowPtrL, &csrColIdxL, &csrValL);
    
    printf("Read matrix done!\n");

    int layer;
    double parallelism;
    int max_row_nnz;
    matrix_layer2<VALUE_TYPE>(m, m, nnzL, csrRowPtrL, csrColIdxL, &layer, &parallelism, &max_row_nnz);

    // x & randomized b
    VALUE_TYPE *x, *b, *x_exact;
    x = (VALUE_TYPE*)malloc(sizeof(VALUE_TYPE) * m);
    b = (VALUE_TYPE*)malloc(sizeof(VALUE_TYPE) * m);
    x_exact = (VALUE_TYPE*)malloc(sizeof(VALUE_TYPE) * m);
    for (int i = 0; i < m; i++)
    {
        x_exact[i] = 1.0;
    }
    get_x_b(m, csrRowPtrL, csrColIdxL, csrValL, x_exact, b);
    free(x_exact);
    // srand(0);
    // for (int i = 0; i < m; i++)
    // {
        // b[i] = rand() * 1.0 / RAND_MAX;
        // b[i] = 1.0;
    // }

    printf("matrix information: location %s\n"
        "m %d nnz %d layer %d parallelism %.2f max_row_nnz %d\n", 
        input_name, m, nnzL, layer, parallelism, max_row_nnz);

    gettimeofday(&tv_begin, NULL);

    PREPROCESSING_STRATEGY ps = ROW_BLOCK;
    SCHEDULE_STRATEGY strategy = SIMPLE;
    int rb = 1;

    int graph_reorder = 0;

    ptr_handler handler;

    if (graph_reorder)
    {
        printf("Begin reordering\n");

        handler = SpTRSV_preprocessing(m, nnzL, csrRowPtrL, csrColIdxL,
        ROW_BLOCK, 1);

        graph_reorder_with_level(handler);

        int permutation[m];

        matrix_reorder(handler, permutation, csrRowPtrL, csrColIdxL, csrValL);

        graph_finalize(handler);
    }

    int flag;
    float sptrsv_time = 0;

    flag = 1;

    // matrix examples
    // atmosmodd
    //anaparas paras = anaparas(1024, 4, ROW_BLOCK, 1, LEVEL_WISE, ONE_LEVEL, SIMPLE, RG_SIMPLE);

    // Wiki-Talk
    //anaparas paras = anaparas(64, 1, ROW_BLOCK, 1, LEVEL_WISE, ONE_LEVEL, WORKLOAD_BALANCE, RG_SIMPLE);

    // cant
    // anaparas paras = anaparas(64, 2, ROW_BLOCK, 1, LEVEL_WISE, ONE_LEVEL, SIMPLE2, RG_SIMPLE);

    // delaunay_n23
    //anaparas paras = anaparas(1024, 4, ROW_BLOCK_AVG, 2, LEVEL_WISE, THRESH_LEVEL, SIMPLE, RG_SIMPLE);

    // delaunay_n13
    // anaparas paras = anaparas(64, 4, ROW_BLOCK_AVG, 1, LEVEL_WISE, THRESH_LEVEL, WORKLOAD_BALANCE, RG_SIMPLE);

    // webbase-1M
    // anaparas paras = anaparas(1024, 1, ROW_BLOCK_THRESH, 32, LEVEL_WISE, ONE_LEVEL, SIMPLE, RG_SIMPLE);

    // my 384**3 d3n4 stencilstar with diag
    // tbs 1024 sws 4 ps 0 8 0 ss 1 0 1
    // 前面都按照顺序，最后三个的顺序是反的
    anaparas paras = anaparas(1024, 4, ROW_BLOCK, 8, LEVEL_WISE, ONE_LEVEL, SIMPLE, RG_BALANCE);

    // my 320**3 d3n14 stencilbox with diag
    // tbs 1024 sws 4 ps 0 4 0 ss 0 1 1
    // anaparas paras = anaparas(1024, 4, ROW_BLOCK, 4, LEVEL_WISE, ONE_LEVEL, SIMPLE2, RG_SIMPLE);

    // my 384**3 d3n7 stencilstar width = 2 with diag
    // tbs 1024 sws 1 ps 1 16 0 ss 0 0 1
    // anaparas paras = anaparas(1024, 1, ROW_BLOCK_THRESH, 16, LEVEL_WISE, ONE_LEVEL, SIMPLE, RG_SIMPLE);

    // my 384**3 stencilstar width = 1 fill in level =1 with diag
    // tbs 1024 sws 4 ps 0 8 0 ss 0 0 1
    // anaparas paras = anaparas(1024, 4, ROW_BLOCK, 8, LEVEL_WISE, ONE_LEVEL, SIMPLE, RG_SIMPLE);

    show_paras(paras);

    ptr_anainfo ana = new anainfo(m);
    SpTRSV_preprocessing_new(m, nnzL, csrRowPtrL, csrColIdxL, ana, paras);

    gettimeofday(&tv_end, NULL);

    printf("Preprocessing time: %.2f us\n", duration(tv_begin, tv_end));

    // copy matrix and vector from CPU to GPU memory
    int *csrRowPtr_d, *csrColIdx_d;
    VALUE_TYPE *csrValue_d, *b_d, *x_d;
    hipMalloc(&csrRowPtr_d, sizeof(int) * (m + 1));
    hipMemcpy(csrRowPtr_d, csrRowPtrL, sizeof(int) * (m + 1), hipMemcpyHostToDevice);
    hipMalloc(&csrColIdx_d, sizeof(int) * nnzL);
    hipMemcpy(csrColIdx_d, csrColIdxL, sizeof(int) * nnzL, hipMemcpyHostToDevice);
    hipMalloc(&csrValue_d, sizeof(VALUE_TYPE) * nnzL);
    hipMemcpy(csrValue_d, csrValL, sizeof(VALUE_TYPE) * nnzL, hipMemcpyHostToDevice);
    hipMalloc(&b_d, sizeof(VALUE_TYPE) * m);
    hipMemcpy(b_d, b, sizeof(VALUE_TYPE) * m, hipMemcpyHostToDevice);
    hipMalloc(&x_d, sizeof(VALUE_TYPE) * m);
    hipMemset(x_d, 0, sizeof(VALUE_TYPE) * m);

    for (int i = 0; i < REPEAT_TIME; i++)
    {
        hipMemset(ana->get_value, 0, sizeof(int) * m);
        hipMemset(x_d, 0, sizeof(VALUE_TYPE) * m);

        hipDeviceSynchronize();

        gettimeofday(&tv_begin, NULL);
        
        SpTRSV_executor_variant(ana, paras, csrRowPtr_d, csrColIdx_d, csrValue_d, b_d, x_d);
        hipDeviceSynchronize();

        gettimeofday(&tv_end, NULL);

        if (i >= WARM_UP) sptrsv_time += duration(tv_begin, tv_end);
    }

    sptrsv_time /= (REPEAT_TIME - WARM_UP);

    hipMemcpy(x, x_d, sizeof(VALUE_TYPE) * m, hipMemcpyDeviceToHost);

    VALUE_TYPE *b_base;
    b_base = (VALUE_TYPE*)malloc(sizeof(VALUE_TYPE) * m);

    get_x_b(m, csrRowPtrL, csrColIdxL, csrValL, x, b_base);

    printf("Solve time: %.2f us\n", sptrsv_time);

    int maxi = error_detect(b, b_base, m);
    VALUE_TYPE error_max = fabs(b[maxi] - b_base[maxi]);
    if (error_max >= ERROR_THRESH)
        printf("Backward max error at index %d, b = %.8f, b_base = %.8f!\n", maxi, b[maxi], b_base[maxi]);
    else
        printf("AG-SpTRSV correct!\n");

    VALUE_TYPE *x_base;
    x_base = (VALUE_TYPE*)malloc(sizeof(VALUE_TYPE) * m);

    #define G (1024 * 1024 * 1024)
    #define M (1024 * 1024)

    float gflops = 1.0 * (2L * nnzL + m) / G;
    // csrValue + ColIdx + x + b + RowPtr
    float gmems = 1.0 * (1L * nnzL * (sizeof(int) + sizeof(VALUE_TYPE)) + 
    2L * m * sizeof(VALUE_TYPE) + 1L * m * sizeof(int)) / G;

    if (outcsv_flag)
    {
        // Write to batch log
        int table_head = 0;
        if (access(outcsv_name, F_OK)) table_head = 1;

        FILE *fp_out;
        fp_out = fopen(outcsv_name, "a");

        if (table_head) fprintf(fp_out, "matrix,m,nnz,layer,parallelism,"
        "sptrsv time(us),sptrsv gflops,sptrsv memory\n");
        fprintf(fp_out, "%s,%d,%d,%d,%.2f,", input_name, m, nnzL, layer, parallelism);

        fprintf(fp_out, "%.2f,%.2f,%.2f\n", sptrsv_time, gflops / sptrsv_time * M, gmems / sptrsv_time * M);
    }

    printf("Gflops: %.4f \nBwidth: %.4f\n", gflops / sptrsv_time * M, gmems / sptrsv_time * M);

    #undef G
    #undef M

    // Finalize
    hipFree(csrRowPtr_d);
    hipFree(csrColIdx_d);
    hipFree(csrValue_d);
    hipFree(x_d);
    hipFree(b_d);

}
